#include <iostream>
#include <memory>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include <hipblas.h>
using namespace std;


// template <typename T>
// __global__ void index_kernel(T *p_index, T *p_input, T *p_output, 
//                              size_t pitch, size_t width, size_t height) 
// { 
//     int   ix = blockDim.x * blockIdx.x + threadIdx.x;
//     int   iy = blockDim.y * blockIdx.y + threadIdx.y;
//     int   tid    = iy * 128 + ix;
//     int   tid_x  = tid;
//     int   tid_y  = iy * 128 + ix;
    
//     // int  *p_src = reinterpret_cast<int *>(p_input);
//     // int  *p_idx = reinterpret_cast<int *>(p_index);
//     // int  *p_dst = reinterpret_cast<int *>(p_output);

//     // for (int i = tid; i < (width>>2) * height; i += (width>>2)) 
//     // if (ix < (width))
//     if (ix < width & iy < height)
//     {
//         // int4 idx_i4 = p_idx[tid];
//         // int4 out_i4;
//         // int  *idx_i = reinterpret_cast<int *>(&idx_i4);
//         // int  *out_i = reinterpret_cast<int *>(&out_i4);

//         // out_i[0] = p_src[0];
//         // out_i[1] = p_src[1];
//         // out_i[2] = p_src[2];
//         // out_i[3] = p_src[3];     
//         // p_dst[tid] = out_i4;
//         // int *p_src = &(p_input[tid]);
//         int idx    = p_index[tid];
//         int *p_src = &(p_input[tid]);
//         p_output[tid] = p_src[idx];//p_input[idx];
//         printf("tid : %d  \tidx : %d  \t p_input: %d  \tp_src : %d\n",
//                tid, idx, p_input[tid_y], p_src[idx]);
//     }
// }
template <typename T>
__global__ void index_kernel(T *p_index, T *p_input, T *p_output, 
                             size_t pitch, size_t width, size_t height) 
{ 
    int   ix = blockDim.x * blockIdx.x + threadIdx.x;
    int   iy = blockDim.y * blockIdx.y + threadIdx.y;
    int   tid    = iy * 128 + ix;
    int   tid_x  = tid;
    int   tid_y  = iy * width + ix;
    
    if (ix < width & iy < height)
    {
        int idx         = p_index[tid];
        p_output[tid_y] = p_input[tid - ix + idx];
    }
}


#define T int
#ifndef ALIGNED
#define ALIGNED(val, align_)  ((val + align_ - 1) & (~(align_ - 1)))
#endif
#define CHECK_ERR(condi, val)  do{ \
    if(condi) {\
        fprintf(stderr, "[%s %d]: CUDA Runtime Error: %d\n", __func__, __LINE__, ret);\
        return ret; \
    }\
}while(0);

int main()
{
    int  batch_size = 2;
    int  width_value = 64;
    int  width_index = 16;
    
    std::vector<T> vec_value(batch_size * width_value);
    std::vector<T> vec_output(batch_size * width_index);
    std::vector<T> vec_index(batch_size * width_index);
    
    for(int i=0; i< batch_size; ++i)
    {
        auto tmp = &(vec_value[i * width_value]);
        for(int j=0; j<width_value; ++j) {
            tmp[j] = i + j * (i + 1);
        }
    }
    for(int i=0; i< batch_size; ++i)
    {
        auto tmp = &(vec_index[i * width_index]);
        for(int j=0; j< width_index; ++j) {
            tmp[j] = j * i + j * j + 1;
        }
    }
    for(int i=0; i< batch_size; ++i)
    {
        std::cout<< ">>> ";
        auto tmp = &(vec_value[i * width_value]);
        for(int j=0; j<width_value; ++j) {
            std::cout << tmp[j] << " " ;
        }
        std::cout<< std::endl;
    }
    std::cout<< std::endl;

    for(int i=0; i< batch_size; ++i)
    {
        auto tmp = &(vec_index[i * width_index]);
        for(int j=0; j< width_index; ++j) {
            std::cout << tmp[j] << " " ;
        }
        std::cout<< std::endl;
    }
    std::cout<< std::endl;

    hipStream_t stream_id;
    T *g_input  = nullptr;
    T *g_output = nullptr;

    size_t pitch_input  = 0;
    size_t pitch_output = 0;
    auto width_output = width_index * sizeof(T);
    auto width_input  = (width_value + width_index) * sizeof(T);

    hipError_t ret = (hipError_t)0;
    ret = hipStreamCreate(&stream_id);
    CHECK_ERR(ret != (hipError_t)0, ret);

    ret = hipMallocPitch((void **)&(g_input),
                &pitch_input, width_input, batch_size);
    CHECK_ERR(ret != (hipError_t)0, ret);

    ret = hipMalloc((void **)&(g_output), batch_size * width_index * sizeof(T));
    CHECK_ERR(ret != (hipError_t)0, ret);
    
    // cout <<  "width_input  : " << width_input << endl <<  "width_output : " << width_output << endl;
    // cout <<  "pitch_input  : " << pitch_input << endl <<  "pitch_output : " << pitch_output << endl;
    // cout << g_input << endl;

    pitch_input = width_input > pitch_input ? width_input : pitch_input;

    std::vector<T> test(batch_size * (width_value + width_index) );
    std::vector<T> dst(batch_size * (width_index));
    
    ret = hipMemcpy2D(g_input, 
                       pitch_input,
                       static_cast<void*>(vec_index.data()),
                       width_output, 
                       width_output, 
                       batch_size,
                       hipMemcpyHostToDevice);
    CHECK_ERR(ret != (hipError_t)0, ret);

    // cout << (void *)(size_t(g_input) + width_output) << endl;
    ret = hipMemcpy2D((void *)(size_t(g_input) + width_output),
                        pitch_input,
                        static_cast<void*>(vec_value.data()),
                        width_value * sizeof(T), // Pitch of source memory
                        width_value * sizeof(T), 
                        batch_size,
                        hipMemcpyHostToDevice);
    CHECK_ERR(ret != (hipError_t)0, ret);
    
    ret = hipMemcpy2D(static_cast<void*>(test.data()), 
                       width_input,
                       g_input, 
                       pitch_input, // Pitch of source memory
                       width_input, 
                       batch_size,
                       hipMemcpyDeviceToHost);
    CHECK_ERR(ret != (hipError_t)0, ret);

    int thread_x = width_index < 128 ? width_index : 128;
    int block_x  = floor(width_index / (thread_x));

    dim3 block_dim(width_index, width_index);
    dim3 gird_dim( (width_index + block_dim.x - 1) / block_dim.x, 
                   (batch_size  + block_dim.y - 1) / block_dim.y);
    cout << block_x << "\t" << thread_x << "\t" << batch_size << endl;

    // index_kernel <<<gird_dim, block_dim, 0, stream_id >>>(g_input,
    //                                                       ((T *)(size_t(g_input) + width_output)),
    //                                                       g_output,
    //                                                       pitch_input,
    //                                                       width_index,  
    //                                                       batch_size);
    index_kernel <<<gird_dim, block_dim, 0, stream_id >>>(g_input,
                                                          ((T *)(size_t(g_input) + width_output)),
                                                          g_output,
                                                          pitch_input,
                                                          width_index,  
                                                          batch_size);


    ret = hipMemcpy2D(static_cast<void*>(dst.data()), 
                       width_output,
                       g_output, 
                       width_output, // Pitch of source memory
                       width_output, 
                       batch_size,
                       hipMemcpyDeviceToHost);
    CHECK_ERR(ret != (hipError_t)0, ret);
    
    for(int i = 0; i < batch_size * width_index; ++i)
    {   
        cout << dst[i] << " ";
        if (i && (i + 1) % width_index == 0) {
            cout << endl;
        }
    }
    std::cout<< std::endl;

    hipFree(g_input);
    hipFree(g_output);
    
}



