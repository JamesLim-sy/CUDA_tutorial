#include "hip/hip_runtime.h"
#include <iostream>
#include <memory>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
using namespace std;

#define _DEBUG_ 1
#define T       double  
#define IndexT  int

#define CHECK_ERR(condi, val)  do{ \
    if(condi) {\
        fprintf(stderr, "[%s %d]: CUDA Runtime Error: %d\n", __func__, __LINE__, ret);\
        return ret; \
    }\
}while(0);

#define START_PRINT(data, width, height)  do {\
    for(int i = 0; i < height; i++) {  \
        std::cout<< ">>> "; \
        auto tmp = &(data[i * width]); \
        for(int j=0; j< width; ++j) { \
            std::cout << tmp[j] << " " ; \ 
        } \
        std::cout<< std::endl; \
    }\
    std::cout<< std::endl;\
}while(0);



template <typename T_, typename IndexT_=int >
__global__ void index_kernel_1(IndexT_ *p_index, T_ *p_input, T_ *p_output, 
                               size_t pitch_idx,
                               size_t pitch_src, 
                               size_t width_index, 
                               size_t height_index) { 
    int   ix = blockDim.x * blockIdx.x + threadIdx.x;
    int   iy = blockDim.y * blockIdx.y + threadIdx.y;
    int   tid    = iy * pitch_idx + ix;
    int   tid_x  = iy * pitch_src + ix;
    int   tid_y  = iy * width_index + ix;

    if (ix < width_index & iy < height_index)  {
        IndexT_ idx     = p_index[tid];
        p_output[tid_y] = p_input[tid_x - ix + idx];
        #if _DEBUG_
        printf("tid : %d \t tid_y : %d \tp_index : %d \tp_input : %d \t p_output : %d\n",
                tid, tid_y, p_index[tid], p_input[tid_x - ix + idx], p_output[tid_y]);
        #endif
    }
}

template <typename T_, typename IndexT_=int >
__global__ void index_kernel_2(IndexT_ *p_index, T_ *p_input, T_ *p_output);


int main()
{
    int  height = 2;
    int  width_value = 100;
    int  width_index = 16;
    
    hipStream_t stream_id;
    hipError_t ret = (hipError_t)0;
    std::vector<T>       vec_value(height * width_value);
    std::vector<T>      vec_output(height * width_index);
    std::vector<IndexT>  vec_index(height * width_index);
    std::vector<T>         vec_dst(height * width_index);
    
    size_t pitch_input  = 0;
    auto   byte_output = width_index * sizeof(T);
    auto   byte_value  = width_value * sizeof(T);
    auto   byte_index  = width_index * sizeof(IndexT);
    IndexT *g_index  = nullptr;
    T      *g_value  = nullptr;
    T      *g_output = nullptr;
    
    for(int i=0; i< height; ++i)  {
        auto tmp = &(vec_value[i * width_value]);
        for(int j=0; j<width_value; ++j) {tmp[j] = i + j * (i + 2); }
    }
    for(int i=0; i< height; ++i) {
        auto tmp = &(vec_index[i * width_index]);
        for(int j=0; j< width_index; ++j) { tmp[j] = (i + 1) * j + j; }
    }
    START_PRINT(vec_value, width_value, height);
    START_PRINT(vec_index, width_index, height);
    hipStreamCreate(&stream_id);

    int  test_case = 1;
    switch (test_case) {
        case 1: {   // 2D  index sample, while 
            ret = hipMallocPitch(
                (void **)&g_index, &pitch_input, 
                byte_index + byte_value, height);
            g_value = (T *)(size_t(g_index) + byte_index);
            
            ret = hipMalloc((void **)&g_output, height * byte_output);
            ret = hipMemcpy2DAsync(
                (void *)g_index, pitch_input,
                static_cast<void*>(vec_index.data()), byte_index, byte_index, height,
                hipMemcpyHostToDevice, stream_id);
            ret = hipMemcpy2DAsync(
                (void *)(size_t(g_index) + byte_index), pitch_input,
                static_cast<void*>(vec_value.data()), byte_value, byte_value, height,
                hipMemcpyHostToDevice, stream_id);

            dim3 block_dim(width_index, width_index);
            dim3 gird_dim((width_index + block_dim.x - 1) / block_dim.x, 
                              (height  + block_dim.y - 1) / block_dim.y);
            // cout << (void *)(size_t(g_value) + width_output) << endl;
            // int thread_x = width_index < 128 ? width_index : 128;
            // int block_x  = floor(width_index / (thread_x));
            index_kernel_1 <<<gird_dim, block_dim, 0, stream_id>>>(
                                       g_index, g_value, g_output,
                                       pitch_input / sizeof(IndexT), 
                                       pitch_input / sizeof(T), 
                                       width_index,
                                       height);
            ret = hipMemcpyAsync(
                static_cast<void*>(vec_dst.data()), (void *)g_output, height * byte_output, 
                hipMemcpyDeviceToHost, stream_id);
            CHECK_ERR(ret != (hipError_t)0, ret);
        }
        case 2 : {  
            std::vector<T>       vec_value(height * width_value);
            std::vector<T>      vec_output(height * width_index);
            std::vector<IndexT>  vec_index(height * width_index);
            std::vector<T>         vec_dst(height * width_index);    
            ret = hipMalloc((void**)&g_index , height * byte_index);
            ret = hipMalloc((void**)&g_value , height * byte_value);
            ret = hipMalloc((void**)&g_output, height * byte_output);       
            hipMemcpyAsync((void *)g_index, static_cast<void*>(vec_index.data()),
                            height * byte_index, hipMemcpyHostToDevice, stream_id);
            hipMemcpyAsync((void *)g_value, static_cast<void*>(vec_value.data()),
                            height * byte_value, hipMemcpyHostToDevice, stream_id);

            int  block_dim = ;
            int  grid_dim  = ;
            index_kernel_2<<< >>>(g_index, g_value, g_output
                            );                   
            hipMemcpyAsync((void *)g_output, static_cast<void*>(vec_output.data()),
                            height * byte_output, hipMemcpyDeviceToHost, stream_id);
        }
    }
    ret = hipStreamSynchronize(stream_id);
    START_PRINT(vec_dst, width_index, height);
    hipFree(g_index);
    hipFree(g_output);
}