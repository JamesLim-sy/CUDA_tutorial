#include <stdint.h>
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

typedef struct _mem_pointer_ {
    float *p_cpu = NULL;
    float *p_gpu = NULL;
} mem_pointer;


#ifndef CHECK_ERR
#define CHECK_ERR(condi, val) do{  \
    if (condi)  \
    {           \
        cout << "[LINE  FUNC]: " << __LINE__ << "\t" << __func__ << "\terr_val = " << val << endl; \
        return  -5;  \
    }               \
}while(0)
#endif


__global__ void  primitive_add(float *x, float *y, float *z, size_t num)
{
    // blockIdx.x : indicates the horizontal index in grid.
    // blockDim.x : indicates the horizontal total thread numbers inside a block.
    // gridDim.x  : indicates the hotizontal total block  numbers inside  a grid.
    int    i      = threadIdx.x + blockIdx.x * blockDim.x;   
    size_t stride = blockDim.x * gridDim.x ; 
    if (i < num) {
       z[i] = x[i] + y[i];
    }
    __syncthreads();  
}


__global__ void  grid_stride_add(float *x, float *y, float *z, size_t num)
{
    int i   = 0;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;   
    size_t stride = blockDim.x * gridDim.x; 
    for (i = idx; i < num; i += stride){
       z[i] = x[i] + y[i];
    }
}


__global__ void  share_mem_add(float *x, float *y, float *z, size_t num)
{
    // __shared__ float cache[2][256];
    __shared__ float cache[256];

    int s   = 0;
    int tid = threadIdx.x;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    cache[tid] = z[idx];
    for (s = idx; s < num; s += 256){
        cache[tid] += x[idx] + y[idx];
    }
    __syncthreads();
    z[idx] = cache[tid];
}


static int rslt_check_func(void   *p_z, 
                           size_t  nByte,
                           size_t  num, 
                           char   *str,
                           hipEvent_t start,
                           hipEvent_t stop)
{
    int ret = 0; 
    mem_pointer *z = (mem_pointer *)p_z;

    // To sync the mission accomplishement of GPU
    hipMemcpy((void *)(z->p_cpu), (void *)(z->p_gpu), nByte, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);  

    float   max_err  = 0.0f; 
    float   timespan = 0.0f;
    hipEventElapsedTime(&timespan, start, stop);

    for (int i = 0; i< num; ++i){
        max_err = fmax(max_err, fabs((z->p_cpu)[i] - 30.0));
        if (max_err > 0.1f){
            cout << "[ERR] value : " << (z->p_cpu)[i];
            cout << "\tindex : " << i << endl;
            return -10; 
        }
    }
    cout << "\tMax  Err  : " << max_err << "\tFunc Name : " << str;
    cout << "\t TimeSpan : " << timespan << " ms";
    cout << "\t Bandwidth(GB/s): " << (num * sizeof(float) * 3) / timespan / 1e6 << endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);

    ret = (fabs(max_err) > 0.1) ? -5 : 0;
    return ret;
}


static int analysis_grid_block(void  *p_x, 
                               void  *p_y, 
                               void  *p_z,
                               int    grid_num, 
                               int    block_num, 
                               size_t nByte)
{
    int   ret = (hipError_t)0;  // which means success
    int     i = 0;
    size_t  N = nByte / sizeof(float); 

    dim3 blocksize(block_num);
    dim3 gridsize(grid_num);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate( &stop);

    mem_pointer  *x = (mem_pointer *)p_x;
    mem_pointer  *y = (mem_pointer *)p_y;
    mem_pointer  *z = (mem_pointer *)p_z;

    for (i = 0; i < N; ++i)
    {
        x->p_cpu[i] = 10.0f;
        y->p_cpu[i] = 20.0f;
    }
    hipMemcpy((void *)(x->p_gpu), (void *)(x->p_cpu), nByte, hipMemcpyHostToDevice);
    hipMemcpy((void *)(y->p_gpu), (void *)(y->p_cpu), nByte, hipMemcpyHostToDevice); 

    {
        memset(z->p_cpu, nByte, 0);
        hipMemset((void *)z->p_gpu, 0, nByte);
        hipEventRecord(start);
        grid_stride_add <<< gridsize, blocksize >>> (x->p_gpu, y->p_gpu, z->p_gpu, N);
        hipEventRecord(stop);
    }
    ret = rslt_check_func(z, nByte, N, "grid-stride loop", start, stop);
    CHECK_ERR(ret != 0, ret);
    return 0;
}


static int analysis_no_grid_block(void  *p_x, 
                                void  *p_y, 
                                void  *p_z,
                                int    grid_num, 
                                int    block_num, 
                                size_t nByte)
{
    int   ret = (hipError_t)0;  // which means success
    int     i = 0;
    size_t  N = nByte / sizeof(float); 

    dim3 blocksize(block_num);
    dim3 gridsize(grid_num);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate( &stop);

    mem_pointer  *x = (mem_pointer *)p_x;
    mem_pointer  *y = (mem_pointer *)p_y;
    mem_pointer  *z = (mem_pointer *)p_z;

    for (i = 0; i < N; ++i)
    {
        x->p_cpu[i] = 10.0f;
        y->p_cpu[i] = 20.0f;
    }
    hipMemcpy((void *)(x->p_gpu), (void *)(x->p_cpu), nByte, hipMemcpyHostToDevice);
    hipMemcpy((void *)(y->p_gpu), (void *)(y->p_cpu), nByte, hipMemcpyHostToDevice); 

    // primitive add
    {
        memset(z->p_cpu, nByte, 0);
        hipMemset((void *)z->p_gpu, 0, nByte);
        hipEventRecord(start);
        primitive_add <<< gridsize, blocksize >>> (x->p_gpu, y->p_gpu, z->p_gpu, N);
        hipEventRecord(stop);
        ret = rslt_check_func(z, nByte, N, "primitive_add", start, stop);
    }
    CHECK_ERR(ret != 0, ret);

    return 0;
}



static size_t   align_mem_alloc(float  **addr, 
                                int      align,
                                size_t   nByte)
{
    int     ret        = (hipError_t)0;
    float  *p_dst      = NULL;
    size_t  mem_length = nByte + align;

    ret   = hipMalloc((void **)addr, mem_length);
    CHECK_ERR(ret != 0, -5);

    p_dst = (float *)((size_t)(*addr) + align);
    CHECK_ERR(p_dst == NULL, -5);
    
    *addr = (float *)p_dst;
    return ret;
}


int main()
{
    int     i = 0, j = 0;
    size_t  N = 1<<24;
    size_t  nByte = N * sizeof(float);
    int     ret = (hipError_t)0;  // which means success
    mem_pointer  x,y,z;

    const size_t block_num_lo = 32;
    const size_t block_num_up = 2048;
    const size_t grid_num_lo = 1024;
    const size_t grid_num_up = 1024<<7; 
    size_t block_num = 0;
    size_t grid_num  = 0;

    x.p_cpu = (float *)malloc(nByte);
    CHECK_ERR(x.p_cpu == NULL, -5);

    y.p_cpu = (float *)malloc(nByte);
    CHECK_ERR(y.p_cpu == NULL, -5);

    z.p_cpu = (float *)malloc(nByte);
    CHECK_ERR(z.p_cpu == NULL, -5);

#if 0
    ret = cudaMalloc((void **)&(x.p_gpu), nByte);
    CHECK_ERR(ret != 0, ret);

    ret = cudaMalloc((void **)&(y.p_gpu), nByte);
    cout << y.p_gpu << endl;
    CHECK_ERR(ret != 0, ret);

    ret = cudaMalloc((void **)&(z.p_gpu), nByte);
    CHECK_ERR(ret != 0, ret);
    cout << "nByte :" << nByte << endl;

    for (j = grid_num_lo; j < grid_num_up ; j <<= 1) {
        cout << "[GRID NUM] : " << j << endl;
        for (i = block_num_lo; i < block_num_up; i <<= 1){  
            cout << "[BLOCK NUM] : " << i; 
            ret = analysis_grid_block(&x, &y, &z, j, i, nByte);
            CHECK_ERR(ret != 0, -5);
        }
        cout << endl;
    }

    j = 0;
    for (i = block_num_lo; i < block_num_up; i <<= 1){  
        j = N / i; 
        cout << "[GRID  NUM] : " << j;
        cout << "[BLOCK NUM] : " << i; 
        ret = analysis_no_grid_block(&x, &y, &z, j, i, nByte);
        CHECK_ERR(ret != 0, -5);
        cout << endl;
    }
#else 
    const int aligned = 33;
    int offset = 0;
    block_num  = 256;
    grid_num   = 65536;
    for (i = 0; i < aligned; i++) 
    {
        offset = i * sizeof(float);
        ret = align_mem_alloc(&(x.p_gpu), offset, nByte);
        CHECK_ERR(ret != 0, ret);

        ret = hipMalloc((void **)&(y.p_gpu), nByte);
        CHECK_ERR(ret != 0, ret);

        ret = hipMalloc((void **)&(z.p_gpu), nByte);
        CHECK_ERR(ret != 0, ret);
        
        cout << "[x.p_gpu]:   " << x.p_gpu;
        cout << "\t[y.p_gpu]: " << y.p_gpu;
        cout << "\t[z.p_gpu]: " << z.p_gpu << endl;

        ret = analysis_grid_block(&x, &y, &z, grid_num, block_num, nByte);
        CHECK_ERR(ret != 0, -5);
    }
#endif
    hipFree(x.p_gpu);
    hipFree(y.p_gpu);
    hipFree(z.p_gpu);
    free(x.p_cpu);
    free(y.p_cpu);
    free(z.p_cpu);    

    return  0;
}
