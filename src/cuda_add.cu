#include <stdint.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include "hipblas.h"
using namespace std;

typedef struct _mem_pointer_ {
    float *p_cpu = NULL;
    float *p_gpu = NULL;
} mem_pointer;


#ifndef CHECK_ERR
#define CHECK_ERR(condi, val) do{  \
    if (condi)  \
    {           \
        cout << "[LINE  FUNC]: " << __LINE__ << "\t" << __func__ << "\terr_val = " << val << endl; \
        return  -5;  \
    }               \
}while(0)
#endif


__global__ void  primitive_add(float *x, float *y, float *z, size_t num)
{
    // blockIdx.x : indicates the horizontal index in grid.
    // blockDim.x : indicates the horizontal total thread numbers inside a block.
    // gridDim.x  : indicates the hotizontal total block  numbers inside  a grid.
    int    i      = threadIdx.x + blockIdx.x * blockDim.x;   
    size_t stride = blockDim.x * gridDim.x ; 
    if (i < num) {
       z[i] = x[i] + y[i];
    }
    // __syncthreads();  
}


__global__ void  grid_stride_add(float *x, float *y, float *z, size_t num)
{
    int i   = 0;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;   
    size_t stride = blockDim.x * gridDim.x; 
    for (i = idx; i < num; i += stride){
       z[i] = x[i] + y[i];
    }
}


__global__ void  share_mem_add(float *x, float *y, float *z, size_t num)
{
    // __shared__ float cache[2][256];
    __shared__ float cache[256];

    int s   = 0;
    int tid = threadIdx.x;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    cache[tid] = z[idx];
    for (s = idx; s < num; s += 256){
        cache[tid] += x[idx] + y[idx];
    }
    __syncthreads();
    z[idx] = cache[tid];
}


static int rslt_check_func(void   *p_z, 
                           size_t  nByte,
                           size_t  num, 
                           char   *str,
                           hipEvent_t start,
                           hipEvent_t stop)
{
    int ret = 0; 
    float   max_err  = 0.0f; 
    float   timespan = 0.0f;
    mem_pointer *z = (mem_pointer *)p_z;

    // To sync the mission accomplishement of GPU
    hipEventSynchronize(stop);  
    hipEventElapsedTime(&timespan, start, stop);
    
    hipMemcpy((void *)(z->p_cpu), (void *)(z->p_gpu), nByte, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    for (int i = 0; i< num; ++i){
        max_err = fmax(max_err, fabs((z->p_cpu)[i] - 30.0));
        if (max_err > 0.1f){
            cout << "[ERR] value : " << (z->p_cpu)[i];
            cout << "\t[index] : " << i << endl;
            return -10; 
        }
    }
    cout << "\t[Max_Err]: " << max_err << " [FuncName]: " << str;
    cout << "\t[TimeSpan]: " << timespan << " ms";
    cout << "\t[Bwidth(GB/s)]: " << (num * sizeof(float) * 3) / timespan / 1e6 << endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);

    ret = (fabs(max_err) > 0.1) ? -5 : 0;
    return ret;
}


static size_t  mem_alloc_offset(float  **addr, 
                                int      align,
                                size_t   nByte)
{
    int     ret        = (hipError_t)0;
    float  *p_dst      = NULL;
    size_t  mem_length = nByte + align;

    ret   = hipMalloc((void **)addr, mem_length);
    CHECK_ERR(ret != 0, -5);

    p_dst = (float *)((size_t)(*addr) + align);
    CHECK_ERR(p_dst == NULL, -5);
    
    *addr = (float *)p_dst;
    return ret;
}


static int analysis_grid_block(void  *p_x, 
                               void  *p_y, 
                               void  *p_z,
                               int    grid_num, 
                               int    block_num, 
                               size_t nByte)
{
    int   ret = (hipError_t)0;  // which means success
    int     i = 0;
    size_t  N = nByte / sizeof(float); 

    dim3 blocksize(block_num);
    dim3 gridsize(grid_num);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate( &stop);

    mem_pointer  *x = (mem_pointer *)p_x;
    mem_pointer  *y = (mem_pointer *)p_y;
    mem_pointer  *z = (mem_pointer *)p_z;

    for (i = 0; i < N; ++i)
    {
        x->p_cpu[i] = 10.0f;
        y->p_cpu[i] = 20.0f;
    }
    hipMemcpy((void *)(x->p_gpu), (void *)(x->p_cpu), nByte, hipMemcpyHostToDevice);
    hipMemcpy((void *)(y->p_gpu), (void *)(y->p_cpu), nByte, hipMemcpyHostToDevice); 

    // if (0)
    // {
    //     memset(z->p_cpu, nByte, 0);
    //     cudaMemset((void *)z->p_gpu, 0, nByte);
    //     cudaEventRecord(start);
    //     primitive_add <<< gridsize, blocksize >>> (x->p_gpu, y->p_gpu, z->p_gpu, N);
    //     cudaEventRecord(stop);
    //     ret = rslt_check_func(z, nByte, N, "primitive_add", start, stop);
    // }
    // else
    // {
    //     memset(z->p_cpu, nByte, 0);
    //     cudaMemset((void *)z->p_gpu, 0, nByte);
    //     cudaEventRecord(start);
    //     grid_stride_add <<< gridsize, blocksize >>> (x->p_gpu, y->p_gpu, z->p_gpu, N);
    //     cudaEventRecord(stop);
    // }
    // ret = rslt_check_func(z, nByte, N, "grid-stride loop", start, stop);
    // CHECK_ERR(ret != 0, ret);

    {
        hipblasHandle_t hdl;
        float alpha = 1.0f;
        hipblasCreate(&hdl);
        memset(z->p_cpu, nByte, 0);
        hipMemset((void *)z->p_gpu, 0, nByte);
        
        hipEventRecord(start);
        ret  = hipblasSaxpy(hdl, N, &alpha, x->p_gpu, 1, y->p_gpu, 1);
        hipEventRecord(stop);

        ret = rslt_check_func(y, nByte, N, "cuBlasSaxpy", start, stop);
        CHECK_ERR(ret != 0, ret);    
        
        ret = hipblasDestroy(hdl);
        CHECK_ERR(ret != 0, ret);
    }
    return 0;
}



int main(int argc, char *argv[])
{
    int     ret = (hipError_t)0;  // which means success
    size_t  N = 1<<24;
    size_t  nByte = N * sizeof(float);
    size_t block_num = 0;
    size_t grid_num  = 0;
    int     i = 0, j = 0, init = 0;
    mem_pointer  x,y,z;
    
    const size_t block_num_lo = 32;
    const size_t block_num_up = 2048;
    const size_t grid_num_lo = 1024;
    const size_t grid_num_up = 1024<<7; 

    if (argc > 1) {
        init = atoi(argv[1]);
        cout << "init : " << init << endl;
    }
    x.p_cpu = (float *)malloc(nByte);
    CHECK_ERR(x.p_cpu == NULL, -5);

    y.p_cpu = (float *)malloc(nByte);
    CHECK_ERR(y.p_cpu == NULL, -5);

    z.p_cpu = (float *)malloc(nByte);
    CHECK_ERR(z.p_cpu == NULL, -5);

    switch  (init) {
        case 0:
        {
            ret = hipMalloc((void **)&(x.p_gpu), nByte);
            CHECK_ERR(ret != 0, ret);
        
            ret = hipMalloc((void **)&(y.p_gpu), nByte);
            cout << y.p_gpu << endl;
            CHECK_ERR(ret != 0, ret);
        
            ret = hipMalloc((void **)&(z.p_gpu), nByte);
            CHECK_ERR(ret != 0, ret);
            cout << "nByte :" << nByte << endl;
        
            for (j = grid_num_lo; j < grid_num_up ; j <<= 1) {
                cout << "[GRID NUM] : " << j << endl;
                for (i = block_num_lo; i < block_num_up; i <<= 1){  
                    cout << "[BLOCK NUM] : " << i; 
                    ret = analysis_grid_block(&x, &y, &z, j, i, nByte);
                    CHECK_ERR(ret != 0, -5);
                }
                cout << endl;
            }
            j = 0;
            for (i = block_num_lo; i < block_num_up; i <<= 1){  
                j = N / i; 
                cout << "[GRID  NUM] : " << j;
                cout << "[BLOCK NUM] : " << i; 
                ret = analysis_grid_block(&x, &y, &z, j, i, nByte);
                CHECK_ERR(ret != 0, -5);
                cout << endl;
            }
            break;
        }

        case 1: {
            const int aligned = 33;
            int offset = 0;
            
            block_num  = 256;
            grid_num   = 65536;
            for (i = 0; i < aligned; i++) 
            {
                offset = i * sizeof(float);
                ret = mem_alloc_offset(&(z.p_gpu), offset, nByte);
                CHECK_ERR(ret != 0, ret);
        
                ret = hipMalloc((void **)&(y.p_gpu), nByte);
                CHECK_ERR(ret != 0, ret);
        
                ret = hipMalloc((void **)&(x.p_gpu), nByte);
                CHECK_ERR(ret != 0, ret);
                
                cout << "[x.p_gpu]:" << x.p_gpu;
                cout << "  [y.p_gpu]:" << y.p_gpu;
                cout << "  [z.p_gpu]:" << z.p_gpu;
                cout << "  [offset ]:" << offset << " "; 
        
                ret = analysis_grid_block(&x, &y, &z, grid_num, block_num, nByte);
                CHECK_ERR(ret != 0, -5);
            }
            break;
        }
        case 2: {
            hipblasStatus_t cu_ret = HIPBLAS_STATUS_SUCCESS;
            ret = hipMalloc((void **)&(z.p_gpu), nByte);
            CHECK_ERR(ret != 0, ret);
    
            ret = hipMalloc((void **)&(y.p_gpu), nByte);
            CHECK_ERR(ret != 0, ret);
    
            ret = hipMalloc((void **)&(x.p_gpu), nByte);
            CHECK_ERR(ret != 0, ret);

            ret = analysis_grid_block(&x, &y, &z, grid_num, block_num, nByte);
            CHECK_ERR(ret != 0, -5);
            break;
        }
        case 3 : {

            break;
        }
        default :{
            cout << "Please select one operation test !" << endl;
            return -5;
        }
    }

    hipFree(x.p_gpu);
    hipFree(y.p_gpu);
    hipFree(z.p_gpu);
    free(x.p_cpu);
    free(y.p_cpu);
    free(z.p_cpu);    
    return  0;
}
