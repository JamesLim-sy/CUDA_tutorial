#include "hip/hip_runtime.h"
#include "cuda_test.h"

/*************************************
* CUDA result check.
*************************************/
template<typename T>
int  rslt_check_func(void   *ptr, 
                     size_t  byte_num,
                     size_t  num, 
                     char   *str,
                     hipEvent_t start,
                     hipEvent_t stop,
                     size_t  loop)
{
    // To sync the mission accomplishement of GPU
    float   timespan = 0.0;
    hipEventSynchronize(stop);  
    hipEventElapsedTime(&timespan, start, stop);
    int     i   = 0 ;
    int     ret = 0; 
    float   eps = 1e-5;
    float   max_err  = 0.0; 
    float   target   = 30.0;
    struct mem_pointer<T> *p_dst = (mem_pointer<T> *)ptr;
    double  val_sum = 0;

    hipMemcpy((void *)(p_dst->p_cpu), (void *)(p_dst->p_gpu), byte_num, hipMemcpyDeviceToHost);

    for (i = 0; i < num; ++i){
        max_err =  __half2float(p_dst->p_cpu[i]) -  __half2float(target);
        if (fabs(max_err) > eps ) {
            cout << "[ERR] value: " << __half2float(p_dst->p_cpu[i]);
            cout << "\t[index] : " << i  << "\t[Loop]: " << loop << endl;
            return -10; 
        }
    }
    p_dst->total_time += timespan;
    p_dst->min_time    = fmin(p_dst->min_time, timespan);
    p_dst->max_time    = fmax(p_dst->max_time, timespan);
    ret = (max_err > eps) ? -5 : 0;
    return ret;
}



template<typename T>
int  grid_block_combination(void   *x, 
                            void   *y, 
                            void   *z, 
                            void   *p_grid_block, 
                            size_t  byte_num,
                            int     loop, 
                            int     func_type) 
{
    int  i = 0, j = 0;
    int  ret  = (hipError_t)0;
    block_thread *p_gb = (block_thread *)p_grid_block;
    struct mem_pointer<T> *p_x = (mem_pointer<T> *)x;
    struct mem_pointer<T> *p_y = (mem_pointer<T> *)y;
    struct mem_pointer<T> *p_z = (mem_pointer<T> *)z;

    cout << "half :" << endl; 
    for (i = p_gb->block_lower; i < p_gb->block_upper; i <<= 1)
    {  
        for (j = p_gb->grid_lower; j < p_gb->grid_upper ; j <<= 1) 
        {
            p_z->total_time = 0.0; 
            p_z->max_time   = 0.0; 
            p_z->min_time   = 1000.0;
            cout << "[BLOCK NUM] : " << i << "  [GRID NUM] : " << j; 
            ret = analysis_grid_block<DATA_TYPE>(&x, &y, &z, j, i, byte_num, loop, func_type);
            CHECK_ERR(ret != 0, -5);
            cout << "\t[AVG]:  " << p_z->total_time / loop ;
            cout << "\t[MAX]: " << p_z->max_time << "\t[MIN]: " << p_z->min_time << endl;
        }
        cout << endl;
    }
    return ret;
}


/*
    cout << "half2_float4 :" << endl;
    for (i = block_num_lo; i < block_num_up; i <<= 1){  
        for (j = grid_num_lo; j < grid_num_up ; j <<= 1) { 
            cout << "[BLOCK NUM] : " << i << " [GRID NUM] : " << j;
            z.total_time = 0.0;  z.max_time   = 0.0; z.min_time   = 1000.0;
            ret = analysis_grid_block<DATA_TYPE>(&x, &y, &z, j, i, byte_num, loop, 3);
            CHECK_ERR(ret != 0, -5);
            cout << "\t[AVG]:  " << z.total_time / loop  << "\t[MAX]: " << z.max_time << "\t[MIN]: " << z.min_time << endl;
        }
        cout << endl;
    }
*/


template<typename T>
int analysis_grid_block(void  *p_x, 
                        void  *p_y, 
                        void  *p_z,
                        void  *p_grid_block,
                        size_t byte_num,
                        int    loop,
                        int    type)
{
    int   ret = (hipError_t)0;  // which means success
    int     i = 0;
    int     j = 0;
    size_t  N = byte_num / sizeof(T);
    float   alpha = 1.0;
    
    hipblasHandle_t hdl;
    hipEvent_t start, stop;
    struct mem_pointer<T> *x  = (mem_pointer<T> *)p_x;
    struct mem_pointer<T> *y  = (mem_pointer<T> *)p_y;
    struct mem_pointer<T> *z  = (mem_pointer<T> *)p_z;
    struct block_thread *p_gb = (block_thread *)p_grid_block;
    
    dim3 blocksize(p_gb->block_num);
    dim3 gridsize(p_gb->grid_num);
    primitive_add <<< p_gb->grid_num, p_gb->block_num >>> (x->p_gpu, y->p_gpu, z->p_gpu, N>>4);  // warm up
   
    while (j < loop) {
        hipEventCreate(&start);
        hipEventCreate( &stop);
        hipMemset((void *)(z->p_gpu), 0, byte_num);

        switch (type) {
            case 0 : {
                hipEventRecord(start);
                grid_stride_add_half <<< p_gb->grid_num, p_gb->block_num >>> (x->p_gpu, y->p_gpu, z->p_gpu, N);
                hipEventRecord(stop);
                break;
            }
            case 1 : {
                hipEventRecord(start);
                grid_stride_add_half2 <<< p_gb->grid_num, p_gb->block_num >>> (x->p_gpu, y->p_gpu, z->p_gpu, N);
                hipEventRecord(stop);
                break;
            }
            case 2 : {
                hipEventRecord(start);
                grid_stride_add_half2_vec2_ld <<< p_gb->grid_num, p_gb->block_num >>> (x->p_gpu, y->p_gpu, z->p_gpu, N);
                hipEventRecord(stop);
                break;
            }
            case 3 : {
                hipEventRecord(start);
                grid_stride_add_half2_vec4_ld <<< p_gb->grid_num, p_gb->block_num >>> (x->p_gpu, y->p_gpu, z->p_gpu, N);
                hipEventRecord(stop);
                break;
            }
            case 4 : {
                hipblasCreate(&hdl); 
                hipEventRecord(start);
                ret  = hipblasSaxpy(hdl, N, &alpha, (float *)x->p_gpu, 1, (float *)y->p_gpu, 1);
                hipEventRecord(stop);
                CHECK_ERR(ret != 0, ret);
                
                hipMemcpy((void *)(z->p_gpu), (void *)(y->p_gpu), byte_num, hipMemcpyDeviceToDevice);
                hipMemcpy((void *)(y->p_gpu), (void *)(y->p_cpu), byte_num, hipMemcpyHostToDevice); 
                hipblasDestroy(hdl);
                break;
            }
            default : {
                return -10;
                break;
            }
        }
        ret = rslt_check_func<T>(z, byte_num, N, "test", start, stop, loop);
        CHECK_ERR(ret != 0, ret);

        hipEventDestroy(start);
        hipEventDestroy(stop); 
        j++;
    }
    return 0;
}


/*************************************
* CUDA memory offset acquire.
*************************************/
template<typename T>
void  mem_alloc_offset(void    *p_src, 
                       int      align,
                       size_t   byte_num)
    struct mem_pointer<T> *p_src = (mem_pointer<T> *)ptr;
    size_t gpu_length = byte_num + align;

    p_src->p_cpu = (DATA_TYPE *)malloc(byte_num);
    hipMalloc((void **)&(p_src->p_gpu_real), gpu_length); 
    p_src->p_gpu = p_src->p_gpu_real;
}


template<typename T>
void mem_alloc(void *ptr, size_t byte_num) 
{
    struct mem_pointer<T> *p_dst = (mem_pointer<T> *)ptr;
    p_dst->p_cpu = (DATA_TYPE *)malloc(byte_num);
    hipMalloc((void **)&(p_dst->p_gpu), byte_num); 
    p_dst->p_gpu_real = nullptr;
}


template<typename T>
void mem_free(void *ptr) 
{
    struct mem_pointer<T> *p  = (mem_pointer<T> *)ptr;
    free(p->p_cpu);

    if (p->p_gpu_real) {
        hipFree(p->p_gpu_real);
    }
    else {
        hipFree(p->p_gpu);
    }
}

